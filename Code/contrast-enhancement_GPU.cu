#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
extern "C"{
#include "hist-equ.h"
}
#include "hist-equ.cu.h"

void free_memory(PGM_IMG *img1, PGM_IMG *img2, PGM_IMG *img3, PGM_IMG *img4, int *img5) {
	free_pgm(*img1);
	free_pgm(*img2);
	hipFree(img3->img);
	hipFree(img4->img);
	hipFree(img5);
}

PGM_IMG contrast_enhancement_g_GPU(PGM_IMG h_img_in)
{
    PGM_IMG d_img_in, h_result, d_result;
    int *d_hist, *d_lut;
    //times
    struct timespec time_start, time_end;

    /*int h_hist[256];
    int hist_CPU[256];
    PGM_IMG result_CPU;
    result_CPU.w = h_img_in.w;
    result_CPU.h = h_img_in.h;
    result_CPU.img = (unsigned char *)malloc(h_img_in.w * h_img_in.h * sizeof(unsigned char));
    if (result_CPU.img == NULL) {
    	printf("%s, line: %d, hipMalloc failed\n", __FILE__, __LINE__);
    	free_pgm(result_CPU);
    	free_pgm(h_img_in);
    	hipDeviceReset();
		exit(1);
    }*/
    
    //Allocate host memory
    h_result.w = h_img_in.w;
    h_result.h = h_img_in.h;
    h_result.img = (unsigned char *)malloc(h_img_in.w * h_img_in.h * sizeof(unsigned char));
    //Allocate device memory
    hipMalloc((void**)&d_img_in.img, h_img_in.w * h_img_in.h * sizeof(unsigned char));
    hipMalloc((void**)&d_result.img, h_img_in.w * h_img_in.h * sizeof(unsigned char));
    hipMalloc((void**)&d_hist, 256 * sizeof(int));
    hipMalloc((void**)&d_lut, 256 * sizeof(int));
    if (d_img_in.img == NULL || d_result.img == NULL || d_hist == NULL || d_lut == NULL) {
    	printf("%s, line: %d, hipMalloc failed\n", __FILE__, __LINE__);
    	free_memory(&h_img_in, &h_result, &d_img_in, &d_result, d_hist);
    	hipDeviceReset();
		exit(1);
    }
    //Initialise device memory
    hipMemcpy(d_img_in.img, h_img_in.img, h_img_in.w * h_img_in.h * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemset(d_hist, 0, 256 * sizeof(int));
    
    /*----------GPU COMPUTATION----------*/

    //time histogram
    clock_gettime(CLOCK_MONOTONIC_RAW, &time_start);
    //Kernel with threads equal to pixels + the extra padding
    histogram_GPU<<<h_img_in.w * h_img_in.h / MAX_THREAD_IN_BLOCK + 1, MAX_THREAD_IN_BLOCK, 256 * sizeof(int)>>>
    	(d_hist, d_img_in.img, h_img_in.h * h_img_in.w, 256);
    clock_gettime(CLOCK_MONOTONIC_RAW, &time_end);
    cudaCheckError();
    printf ("GPU time = %15.10f seconds\n", time_format(time_start, time_end));
    
    //hipMemcpy(h_hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);
    //printf("GPU HISTOGRAM\n\n");
    //print_histogram(h_hist, 256);
    //histogram(hist_CPU, h_img_in.img, h_img_in.h * h_img_in.w, 256);
    //printf("CPU HISTOGRAM\n\n");
    //print_histogram(hist_CPU, 256);
    //histogram_diff(h_hist, hist_CPU, 256);

    //time histogram equalization
    //clock_gettime(CLOCK_MONOTONIC_RAW, &time_start);
    histogram_lut_GPU<<<1, 1>>>
    	(d_hist, d_lut, h_img_in.w * h_img_in.h, 256);
    cudaCheckError();
    histogram_equalization_GPU<<<h_img_in.w * h_img_in.h / MAX_THREAD_IN_BLOCK + 1, MAX_THREAD_IN_BLOCK>>>
    	(d_result.img, d_img_in.img, d_lut, h_img_in.w * h_img_in.h);
    cudaCheckError();
    //clock_gettime(CLOCK_MONOTONIC_RAW, &time_end);
    //printf ("GPU time = %15.10f seconds\n", time_format(time_start, time_end));
    hipMemcpy(h_result.img, d_result.img, h_img_in.w * h_img_in.h * sizeof(unsigned char), hipMemcpyDeviceToHost);
    //CPU histogram equalization and diff
    //histogram_equalization(result_CPU.img, h_img_in.img, hist_CPU, h_img_in.w * h_img_in.h, 256);
    //img_diff(result_CPU, h_result);
    
    //free_pgm(result_CPU);
    hipFree(d_img_in.img);
    hipFree(d_result.img);
    hipFree(d_hist);
    hipDeviceReset();
    return h_result;
}
