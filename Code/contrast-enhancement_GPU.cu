#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
extern "C"{
#include "hist-equ.h"
}
#include "hist-equ.cu.h"

void free_memory(PGM_IMG *img1, PGM_IMG *img2, PGM_IMG *img3, PGM_IMG *img4, int *img5) {
	free_pgm(*img1);
	free_pgm(*img2);
	hipFree(img3->img);
	hipFree(img4->img);
	hipFree(img5);
}

PGM_IMG contrast_enhancement_g_GPU(PGM_IMG img_in)
{
    PGM_IMG d_img_in, result, d_result;
    int hist[256], *d_hist;
    //times
    struct timespec time_start, time_end;

    int hist_CPU[256];
    
    //Allocate host memory
    result.w = img_in.w;
    result.h = img_in.h;
    result.img = (unsigned char *)malloc(result.w * result.h * sizeof(unsigned char));
    //Allocate device memory
    hipMalloc((void**)&d_img_in.img, img_in.w * img_in.h * sizeof(unsigned char));
    hipMalloc((void**)&d_result.img, img_in.w * img_in.h * sizeof(unsigned char));
    hipMalloc((void**)&d_hist, 256 * sizeof(int));
    if (d_img_in.img == NULL || d_result.img == NULL || d_hist == NULL) {
    	printf("%s, line: %d, hipMalloc failed\n", __FILE__, __LINE__);
    	free_memory(&img_in, &result, &d_img_in, &d_result, d_hist);
    	hipDeviceReset();
		exit(-1);
    }
    //Initialise device memory
    hipMemcpy(d_img_in.img, img_in.img, img_in.w * img_in.h * sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemset(d_hist, 0, 256 * sizeof(int));
    
    /*----------GPU COMPUTATION----------*/

    //time histogram
    clock_gettime(CLOCK_MONOTONIC_RAW, &time_start);
    //Kernel with threads equal to pixels + the extra padding
    histogram_GPU<<<img_in.w * img_in.h / MAX_THREAD_IN_BLOCK + 1, MAX_THREAD_IN_BLOCK>>>
    	(d_hist, d_img_in.img, img_in.h * img_in.w, 256);
    clock_gettime(CLOCK_MONOTONIC_RAW, &time_end);
    cudaCheckError();
    printf ("GPU time = %15.10f seconds\n", time_format(time_start, time_end));
    //printf("GPU HISTOGRAM\n\n");
    //print_histogram(hist, 256);
    //CPU histogram and diff
    //histogram(hist_CPU, img_in.img, img_in.h * img_in.w, 256);
    //histogram_diff(hist, hist_CPU, 256);


    histogram_equalization_GPU<<<1, 4>>>(d_result.img,d_img_in.img,hist,result.w*result.h, 256);

    hipMemcpy(hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceReset();
    return result;
}
