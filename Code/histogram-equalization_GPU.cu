#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.cu.h"

extern "C"{
#include "hist-equ.h"
}

__device__ void print_id() {
    printf("by: %d, bx: %d, tx: %d, ty: %d\n", blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x);
    return;
}

__global__ void histogram_GPU(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin) {

    int pixel;

    //if (((blockIdx.x / blockDim.x) + 1) * (blockDim.x * blockDim.x) < img_size) {
        //pixel = (blockIdx.x / blockDim.x) * (blockDim.x * blockDim.x) + (threadIdx.x * blockDim.x) + (blockIdx.x % blockDim.x);
    //}
    //else {
        pixel = blockIdx.x * blockDim.x + threadIdx.x;
    //}
    //pixel = threadIdx.x * img_size / MAX_THREAD_IN_BLOCK + blockIdx.x;
    /*if (blockIdx.x == 1024 && threadIdx.x == 0) {
        printf("pixel: %d\n", pixel);
    }*/
    extern __shared__ int sh_hist_out[];
    if (threadIdx.x < nbr_bin) {
        sh_hist_out[threadIdx.x] = 0;
    }

    __syncthreads();

    if (pixel < img_size) {
        atomicAdd(&sh_hist_out[img_in[pixel]], 1);
    }

    __syncthreads();

    if (threadIdx.x < nbr_bin) {
        atomicAdd(&hist_out[threadIdx.x], sh_hist_out[threadIdx.x]);
    }
}

__global__ void histogram_lut_GPU(int * hist_in, int * lut, int img_size, int nbr_bin) {
    int i, cdf, min, d;
    
    /* Construct the LUT by calculating the CDF */
    cdf = 0;
    min = 0;
    i = 0;
    while (min == 0) {
        min = hist_in[i++];
    }
    d = img_size - min;
    for (i = 0; i < nbr_bin; i ++) {
        cdf += hist_in[i];
        //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
        lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
        if (lut[i] < 0) {
            lut[i] = 0;
        }
        else if (lut[i] > 255) {
            lut[i] = 255;
        }
    }
}

__global__ void histogram_equalization_GPU(unsigned char * img_out, unsigned char * img_in, 
                            int * lut, int img_size) {
    
    int pixel;
    pixel = blockIdx.x * blockDim.x + threadIdx.x;

    /* Get the result image */
    if (pixel < img_size) {
        img_out[pixel] = (unsigned char)lut[img_in[pixel]];
    }
}
