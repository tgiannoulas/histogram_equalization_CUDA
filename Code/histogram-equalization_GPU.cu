#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "hist-equ.cu.h"

extern "C"{
#include "hist-equ.h"
}

__device__ void print_id() {
    printf("by: %d, bx: %d, tx: %d, ty: %d\n", blockIdx.y, blockIdx.x, threadIdx.y, threadIdx.x);
    return;
}

__global__ void histogram_GPU(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin) {

    int pixel = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (pixel < img_size) {
        atomicAdd(&hist_out[img_in[pixel]], 1);
    }
}

__global__ void histogram_lut_GPU(int * hist_in, int * lut, int img_size, int nbr_bin) {

    int pixel, i, cdf, min, d;
    pixel = blockIdx.x * blockDim.x + threadIdx.x;

    /* Construct the LUT by calculating the CDF */
    if (pixel == 0) {
        cdf = 0;
        min = 0;
        i = 0;
        while (min == 0) {
            min = hist_in[i++];
        }
        d = img_size - min;
        for (i = 0; i < nbr_bin; i ++) {
            cdf += hist_in[i];
            //lut[i] = (cdf - min)*(nbr_bin - 1)/d;
            lut[i] = (int)(((float)cdf - min)*255/d + 0.5);
            if (lut[i] < 0) {
                lut[i] = 0;
            }
            else if (lut[i] > 255) {
                lut[i] = 255;
            }
        }
    }
}

__global__ void histogram_equalization_GPU(unsigned char * img_out, unsigned char * img_in, 
                            int * lut, int img_size) {
    
    int pixel;
    pixel = blockIdx.x * blockDim.x + threadIdx.x;

    /* Get the result image */
    if (pixel < img_size) {
        img_out[pixel] = (unsigned char)lut[img_in[pixel]];
    }
}
